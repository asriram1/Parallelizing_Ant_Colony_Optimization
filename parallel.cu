#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<math.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand.h>
#include<time.h>


#define MAX_CITIES 29	
#define MAX_ANTS 14		
#define Q 80
#define ALPHA 0.5
#define BETA 0.8 
#define RHO 0.5 

using namespace std;

int n=0;
int NC = 0;
int t = 0;
struct cities
{
	int x,y;
};
int s;
struct ANTS{
	
	int curCity, nextCity;
	int visited[MAX_CITIES];
	int tour[MAX_CITIES];
	float L;
};

cities city[MAX_CITIES];
float pheromone[MAX_CITIES][MAX_CITIES];
float dist[MAX_CITIES][MAX_CITIES];
ANTS ant[MAX_ANTS];
int best=9999999;
int bestIndex;
float Delta_Pheromones[MAX_CITIES][MAX_CITIES];
float numerator[MAX_CITIES][MAX_CITIES];
hiprandState  state[MAX_ANTS];


__global__ void initialize(float *d_dist,float *d_pheromone,float *d_Delta_Pheromones,cities *d_city,int n)
{	



	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;



	if((row<n)&&(col<n)){
	
		d_dist[col + row * n] = 0.0f;
		d_pheromone[col + row * n] = 1.0f / n;
		d_Delta_Pheromones[col + row * n] = 0.0f;
		if(row!=col)
		{
			d_dist[col + row * n]=sqrt(powf(abs(d_city[row].x-d_city[col].x),2)+powf(abs(d_city[row].y-d_city[col].y),2));
			
		}
	}



}


__global__ void setup_curand_states(hiprandState *state_d,int t){
	
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprand_init(t, id, 0, &state_d[id]);
}

__device__ float generate(hiprandState* globalState, int ind){
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}



__global__ void initializeTour(ANTS *d_ant,int n){
	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<n){
		int j = id;
		d_ant[id].curCity = j;
		for(int i=0;i<n;i++)
		{
			d_ant[id].visited[i]=0;
		}
		d_ant[id].visited[j] = 1;
		d_ant[id].tour[0] = j;
		d_ant[id].L = 0.0;
	}
}

__global__ void PHI_numerator(float *d_numerator, float *d_dist, float *pheromone, int n){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < n && col < n){
		int id = row * n + col;
		d_numerator[id] =  powf( pheromone[id], ALPHA) * powf( (1.0/ d_dist[id]), BETA);
	}
}

__device__ int nextCity(int k,int n,float *d_numerator,ANTS *d_ant,hiprandState *state_d)
{	
	int i = d_ant[k].curCity;
	int j;
	double sum=0.0;
	for(j=0;j<n;j++)
	{
		if(d_ant[k].visited[j]==0)
		{
			sum+= d_numerator[i*n+j];
		}
	}
	
	while(1)
	{
		j++;
		if(j >= n)
			j=0;
		if(d_ant[k].visited[j] == 0)
		{
			float probability = d_numerator[i*n+j]/sum;
			float random = (float)generate(state_d,i); 
			
			if(random < probability)
			{
				break;
			}
		}
	}
	
	return j;
}

__global__ void tourConstruction(ANTS *d_ant, float *d_dist, float *d_numerator,int n,hiprandState *state_d)
{	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < n){
		for(int s=1;s<n;s++)
		{	
			int j = nextCity(id, n, d_numerator,d_ant,state_d);	
			d_ant[id].nextCity = j;
			d_ant[id].visited[j]=1;
			d_ant[id].tour[s] = j;			
			d_ant[id].L+=d_dist[d_ant[id].curCity * n + j];
			d_ant[id].curCity = j;
		}
	}
}
__global__
void endTour(float *Delta_Pheromones, ANTS *ant,float *dist, int *best, int *bestIndex){
	
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	if(k < MAX_ANTS){
		ant[k].L += dist[ant[k].curCity * MAX_CITIES + ant[k].tour[0]];
		ant[k].curCity = ant[k].tour[0];
		
		int temp = *best;
		printf("This is before atomicMin %d\n", *best);
		atomicMin(best, ant[k].L);
		printf("This is after atomicMin %d\n", *best);
		if (*best!= temp){
			*bestIndex = k;
		}
		for(int i = 0; i < MAX_CITIES;i++){
			int first = ant[k].tour[i];
			int second = ant[k].tour[(i + 1) % MAX_CITIES];
			Delta_Pheromones[first * MAX_CITIES + second] += Q/ant[k].L;
		}
	}
	
}
__global__ void updatePheromone(float *d_pheromone, float *d_Delta_Pheromones, int n){

	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < n){
		for(int s=0;s<n;s++){
			if(id!=s)
			{
				d_pheromone[id*n+s] *=( 1.0 - RHO);
				
				if(d_pheromone[id*n+s]<0.0)
				{
					d_pheromone[id*n+s] = (1.0/n);
				}
			}
			d_pheromone[id*n+s] += d_Delta_Pheromones[id*n+s];
			d_Delta_Pheromones[id*n+s] = 0;	
		}
	}
}
__global__ void emptyTour(ANTS *d_ant,float *d_Delta_Pheromones,int n){
	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id < n){
		
		for(int s=0;s<n;s++){		
			d_ant[id].tour[s] = 0;
			d_ant[id].visited[s] = 0;
		}	
	}
}

int main(int argc, char *argv[])


{	

	clock_t start = clock();



if (argc > 1){
		cout << "Accessing file "<< argv[1]<<endl;
	}
	else{
		cout << "Input File Name!" <<endl;
		return 1;
	}
	ifstream in;
    	in.open(argv[1]);
	in>>n;
	cout<<n<<endl;
	int num;
	for(int i=0;i<n;i++)
	{
		in>>num;	
		in>>city[i].x;
		in>>city[i].y;
		cout<<city[i].x<<" "<<city[i].y<<" "<<endl;	
	}
	
	dim3 blockDim(32, 32, 1);
	dim3 gridDim((n - 1)/ 32 + 1, (n - 1)/ 32 + 1, 1 );
	float *d_dist,*d_pheromone,*d_Delta_Pheromones,*d_numerator;
	ANTS *d_ant;
	cities *d_city;
	hiprandState  *state_d;
	int *d_best, *d_bestIndex;
	hipMalloc((void**)&d_pheromone, sizeof(float) * n * n);
	hipMalloc((void**)&d_dist, sizeof(float) * n * n);
	hipMalloc((void**)&d_Delta_Pheromones, sizeof(float) * n * n);
	hipMalloc((void**)&d_ant, sizeof(ANTS) * n);
	hipMalloc((void**)&d_city, sizeof(cities) * n);
	hipMalloc((void**)&d_numerator, sizeof(float) * n *n);
	hipMalloc( (void**) &state_d, sizeof(state));
	hipMalloc((void **)&d_best, sizeof(int));
	hipMalloc((void **)&d_bestIndex, sizeof(int));
	hipMemcpy(d_city,city,sizeof(cities) * n,hipMemcpyHostToDevice);
	srand(time(0));
        hipMemcpy(d_best, &best, sizeof(int), hipMemcpyHostToDevice);	
	int seed = rand();
	setup_curand_states <<< (n-1)/32+1,32 >>> (state_d,seed);
	initialize<<<gridDim, blockDim>>>(d_dist,d_pheromone,d_Delta_Pheromones,d_city,n);
	hipMemcpy(dist,d_dist,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	hipMemcpy(pheromone,d_pheromone,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	hipMemcpy(Delta_Pheromones,d_Delta_Pheromones,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	int MAX_TIME = 20;
	for(;;)
	{		
		initializeTour<<<(n-1)/32+1,32>>>(d_ant,n);
		hipDeviceSynchronize();
		PHI_numerator<<< gridDim, blockDim>>>(d_numerator, d_dist, d_pheromone, n);
		hipDeviceSynchronize();
		tourConstruction<<<(n-1)/32+1,32>>>(d_ant,d_dist,d_numerator,n,state_d);
		hipDeviceSynchronize();
		hipMemcpy(ant,d_ant,sizeof(ANTS) * n,hipMemcpyDeviceToHost);
		endTour<<<(n - 1)/32 + 1, 32>>>(d_Delta_Pheromones, d_ant, d_dist, d_best, d_bestIndex);
		updatePheromone<<< (n-1)/32+1,32>>>(d_pheromone,d_Delta_Pheromones,n);
		hipDeviceSynchronize();
		t += MAX_ANTS;
		NC += 1;
		if(NC < MAX_TIME){
			emptyTour<<<(n-1)/32+1,32>>>(d_ant,d_Delta_Pheromones,n);
			hipMemcpy(&best, d_best, sizeof(int), hipMemcpyDeviceToHost);
			cout<<"Best so far = "<<best<<endl;
			hipDeviceSynchronize();
		}
		else{
			break;
		}
	}
	cout<<endl;
	hipMemcpy(&best, d_best, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&bestIndex, d_bestIndex, sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++)
	{
		cout<<ant[bestIndex].tour[i]<<" ";
	}
	cout<<endl;


	cout<<"\n Best tour = "<<best<<endl<<endl<<endl;


	clock_t last = clock();



	cout<< double(last - start) / CLOCKS_PER_SEC <<endl;
	return 0;
}